
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void Factorial(int *gpu_num,long int *gpu_res)
{
  int i;
  *gpu_res=1;
  for(i=1;i<=*gpu_num;i++)
  {
    *gpu_res = *gpu_res * i;      
  }
}

int main()
{
  int Number;  
  int *dev_number;
  long int *res, result; //store result 
  
  printf("\n\t Enter the number : ");
  scanf("%d",&Number);
  
  //to allocate memory for a number on the GPU/Device
  hipMalloc((void**)&dev_number,sizeof(int));
  hipMalloc((void**)&res,sizeof(long int));
  
  //copy number to the GPU/Device memory
  hipMemcpy(dev_number,&Number,sizeof(int),hipMemcpyHostToDevice);
 
  //call square function which will execute parallely on GPU
  Factorial<<<1,1>>>(dev_number,res);

  //copy result back from device/GPU back to CPU/Host
  hipMemcpy(&result,res,sizeof(long int),hipMemcpyDeviceToHost);

  //display result on the screen
  printf("\n\t Factorial of number %d is %ld \n",Number,result); 
 
  //deallocate GPU/Device memory
  return 0; 
}

